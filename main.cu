
#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>

#define N 3

__global__ void inc(int *a)
{
    int i = blockIdx.x;
    if (i < N)
    {
        a[i]++;
    }
}

int main()
{
    int ha[N], *da;
    hipMalloc((void **)&da, N * sizeof(int));
    for (int i = 0; i < N; ++i)
    {
        ha[i] = i;
    }
    hipMemcpy(da, ha, N * sizeof(int), hipMemcpyHostToDevice);
    inc<<<N, 1>>>(da);
    hipMemcpy(ha, da, N * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; ++i)
    {
        assert(ha[i] == i + 1);
    }
    hipFree(da);

    printf("No assert errors.\n");
    return 0;
}
